#include <string.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>


// Size of input data & result buffers
#define BUF 30

/* Function computing the final string to print */
__global__ void ComputeString(char *res, char *a, char *b, char *c, int length)
{
    int i;

    for (i = 0; i < length; i++)
    {
        res[i] = a[i] + b[i] + c[i];
    }
}

int main()
{
    // Arrays declaration in host memory
    char a[BUF] = {40, 70, 70, 70, 80, 0, 50, 80, 80, 70, 70, 0, 40, 80, 79, 70, 0, 40, 50, 50, 0, 70, 80, 0, 30, 50, 30, 30, 0, 0};
    char b[BUF] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10,
                   10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 0, 0};
    char c[BUF] = {22, 21, 28, 28, 21, 22, 27, 21, 24, 28, 20, 22, 20, 24, 22,
                   29, 22, 21, 20, 25, 22, 25, 20, 22, 27, 25, 28, 25, 0, 0};

    // Results array allocation in host memory
    char *res;
    res = (char *)malloc(BUF * sizeof(char));

    // Allocate arrays in device memory
    size_t size = BUF * sizeof(char);

    char *d_a;
    hipMalloc(&d_a, size);
    char *d_b;
    hipMalloc(&d_b, size);
    char *d_c;
    hipMalloc(&d_c, size);
    char *d_res;
    hipMalloc(&d_res, size);

    // Copy array data from host memory to device memory (res is not necessary)
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    // TODO: threads?
    int N = BUF;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    ComputeString<<<blocksPerGrid, threadsPerBlock>>>(d_res, d_a, d_b, d_c, BUF);

    // Copy result from device memory to host memory
    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_res);

    // Report result & free host memory
    printf("%s\n", res);
    free(res);

    return 0;
}
