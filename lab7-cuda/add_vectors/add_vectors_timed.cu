#include <string.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>


// Workarounds to test locally on my machine (windows)
#ifdef _WIN32
#include <Windows.h>
long lrand48();
void srand48(long seedval);
int gettimeofday(struct timeval *tv, struct timezone *tz);
#else
#include <sys/time.h>
#endif

// Device code
__global__ void VecAdd(int *A, int *B, int *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main(int argc, char **argv)
{
    // Take two input parameters (through command line):
    // N for the number of elements in vector and S for the seed used to fill the two input vectors
    int N;
    int S;

    // For time metrics
    struct timeval t1, t2;
    double temp_duration, duration;

    // Check the input arguments
    if (argc < 3)
    {
        printf("Usage: %s S N\n", argv[0]);
        printf("\tS: seed for pseudo-random generator\n");
        printf("\tN: size of the array\n");
        exit(1);
    }

    S = atoi(argv[1]);
    N = atoi(argv[2]);
    srand48(S);

    // Allocate input vectors h_A and h_B in host memory
    size_t size = N * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);

    // Allocate result vector in host memory
    int *h_res = (int *)malloc(size);

    // Initialize input vectors with random integer values (with the seed S)
    for (int i = 0; i < N; i++)
    {
        h_A[i] = lrand48() % N;
        h_B[i] = lrand48() % N;
    }

    // Allocate vectors in device memory
    int *d_A;
    hipMalloc(&d_A, size);
    int *d_B;
    hipMalloc(&d_B, size);
    int *d_res;
    hipMalloc(&d_res, size);

    // Copy vectors from host memory to device memory
    gettimeofday(&t1, NULL);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    gettimeofday(&t2, NULL);
    temp_duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
    duration = temp_duration;
    printf("Transfer Host -> Device took %lf s\n", temp_duration);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    gettimeofday(&t1, NULL);

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_res, N);

    gettimeofday(&t2, NULL);
    temp_duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
    duration += temp_duration;
    printf("Computation in device took %lf s\n", temp_duration);

    // Copy result from device memory to host memory
    gettimeofday(&t1, NULL);

    hipMemcpy(h_res, d_res, size, hipMemcpyDeviceToHost);

    // Calc. total time
    gettimeofday(&t2, NULL);
    temp_duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);
    printf("Transfer Device -> Host took %lf s\n", temp_duration);

    duration += temp_duration;
    printf("Total duration: %lf s\n", duration);

    // Check that the resulting vector C is the sum of A and B
    float error = 0;
    float tol = 1e-6;
    int temp = 0;
    for (int j = 0; j < N; j++)
    {
        temp = h_A[j] + h_B[j];
        error += abs(h_res[j] - temp);
#ifdef DEBUG
        printf("temp = %d + %d = %d | h_res[j] = %d\n", h_A[j], h_B[j], temp, h_res[j]);
#endif
    }
    printf("Accumulated error: %f\n", error);
    printf("Sum is correct: %s\n", abs(error) <= tol ? "True" : "False");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_res);
}

#ifdef _WIN32
long lrand48()
{
    return rand();
}

void srand48(long seedval)
{
    srand(seedval);
}

struct timezone
{
    int tz_minuteswest;
    int tz_dsttime;
};

int gettimeofday(struct timeval *tv, struct timezone *tz)
{
    // Source: https://stackoverflow.com/a/59359900/8522453
    if (tv)
    {
        FILETIME filetime; /* 64-bit value representing the number of 100-nanosecond intervals since January 1, 1601 00:00 UTC */
        ULARGE_INTEGER x;
        ULONGLONG usec;
        static const ULONGLONG epoch_offset_us = 11644473600000000ULL; /* microseconds betweeen Jan 1,1601 and Jan 1,1970 */

#if _WIN32_WINNT >= _WIN32_WINNT_WIN8
        GetSystemTimePreciseAsFileTime(&filetime);
#else
        GetSystemTimeAsFileTime(&filetime);
#endif
        x.LowPart = filetime.dwLowDateTime;
        x.HighPart = filetime.dwHighDateTime;
        usec = x.QuadPart / 10 - epoch_offset_us;
        tv->tv_sec = (time_t)(usec / 1000000ULL);
        tv->tv_usec = (long)(usec % 1000000ULL);
    }
    return 0;
}
#endif